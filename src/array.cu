
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>


// Kernel function to add the elements of two arrays


__global__
void haversine(int n, float *x, float *y)
{


  //int index = threadIdx.x;
  //int stride = blockDim.x;
 // for (int i = index; i < n; i += stride)
  
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  
  float R = 6378.137;
  float toRad = 3.14159/180;
  
  for (int i = index; i < n; i += stride) {
    // y[i] = atan(sqrt(x[i])) * sin(sqrt(y[i])) *4;
    
    float lon1 = x[i];
    float lon2 = x[i];
    float lat1 = y[i];
    float lat2 = y[i];
            
    lon1 = lon1 * toRad;
    lon2 = lon2 * toRad;
    lat1 = lat1 * toRad;
    lat2 = lat2 * toRad;
    float dlon = lon2 - lon1;
    float dlat = lat2 - lat1;
            
    double a = pow(sin(dlat / 2), 2) + (cos(lat1) * cos(lat2) * pow(sin(dlon / 2),2));
    double d = 2 * atan2(sqrt(a), sqrt(1 - a)) * R;
    x[i] = float(d);
    }
}

int main(void)
{
  int N = pow(2,30);
  std::cout << "In: " << N << std::endl;

  float *x, *y;

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  // Run kernel on 1M elements on the GPU
  // int blockSize = 1024;
  // int numBlocks = (N + blockSize - 1) / blockSize;
  // haversine<<<numBlocks, blockSize>>>(N, x, y);
  //add<<<1,1>>>(N, x, y);
  
  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;
  std::cout << numBlocks << std::endl;
  
  for (int z = 1; z < 1000; z++) { // Run 1000 calls to function to fill GPU for ~1 minute 
    haversine<<<numBlocks, blockSize>>>(N, x, y);
  }
  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  std::cout << "First: " << x[0] << std::endl;

  // Free memory
  hipFree(x);
  hipFree(y);
  
  return 0;
}